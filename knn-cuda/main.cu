/** Modifed version of knn-CUDA from https://github.com/vincentfpgarcia/kNN-CUDA
 * The modifications are
 *      removed texture memory usage
 *      removed split query KNN computation
 *      added feature extraction with bilinear interpolation
 *
 * Last modified by Christopher B. Choy <chrischoy@ai.stanford.edu> 12/23/2016
 */

// Includes
#include <cstdio>
#include <algorithm>
#include <sys/time.h>
#include <time.h>
#include <hip/hip_runtime.h>

// Constants used by the program
#define BLOCK_DIM 16


/**
  * Computes the distance between two matrix A (reference points) and
  * B (query points) containing respectively wA and wB points.
  *
  * @param A     pointer on the matrix A
  * @param wA    width of the matrix A = number of points in A
  * @param B     pointer on the matrix B
  * @param wB    width of the matrix B = number of points in B
  * @param dim   dimension of points = height of matrices A and B
  * @param AB    pointer on the matrix containing the wA*wB distances computed
  */
__global__ void cuComputeDistanceGlobal(float *A, int wA, float *B, int wB,
                                        int dim, float *AB) {

  // Declaration of the shared memory arrays As and Bs used to store the
  // sub-matrix of A and B
  __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
  __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

  // Sub-matrix of A (begin, step, end) and Sub-matrix of B (begin, step)
  __shared__ int begin_A;
  __shared__ int begin_B;
  __shared__ int step_A;
  __shared__ int step_B;
  __shared__ int end_A;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Other variables
  float tmp;
  float ssd = 0;

  // Loop parameters
  begin_A = BLOCK_DIM * blockIdx.y;
  begin_B = BLOCK_DIM * blockIdx.x;
  step_A = BLOCK_DIM * wA;
  step_B = BLOCK_DIM * wB;
  end_A = begin_A + (dim - 1) * wA;

  // Conditions
  int cond0 = (begin_A + tx < wA); // used to write in shared memory
  int cond1 = (begin_B + tx < wB); // used to write in shared memory & to
                                   // computations and to write in output matrix
  int cond2 =
      (begin_A + ty < wA); // used to computations and to write in output matrix

  // Loop over all the sub-matrices of A and B required to compute the block
  // sub-matrix
  for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {
    // Load the matrices from device memory to shared memory; each thread loads
    // one element of each matrix
    if (a / wA + ty < dim) {
      shared_A[ty][tx] = (cond0) ? A[a + wA * ty + tx] : 0;
      shared_B[ty][tx] = (cond1) ? B[b + wB * ty + tx] : 0;
    } else {
      shared_A[ty][tx] = 0;
      shared_B[ty][tx] = 0;
    }

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Compute the difference between the two matrixes; each thread computes one
    // element of the block sub-matrix
    if (cond2 && cond1) {
      for (int k = 0; k < BLOCK_DIM; ++k) {
        tmp = shared_A[k][ty] - shared_B[k][tx];
        ssd += tmp * tmp;
      }
    }

    // Synchronize to make sure that the preceding computation is done before
    // loading two new sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory; each thread writes one element
  if (cond2 && cond1)
    AB[(begin_A + ty) * wB + begin_B + tx] = ssd;
}

/**
  * Gathers k-th smallest distances for each column of the distance matrix in
 * the top.
  *
  * @param dist        distance matrix
  * @param ind         index matrix
  * @param width       width of the distance matrix and of the index matrix
  * @param height      height of the distance matrix and of the index matrix
  * @param k           number of neighbors to consider
  */
__global__ void cuInsertionSort(float *dist, int *ind, int width, int height,
                                int k) {

  // Variables
  int l, i, j;
  float *p_dist;
  int *p_ind;
  float curr_dist, max_dist;
  int curr_row, max_row;
  unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

  if (xIndex < width) {
    // Pointer shift, initialization, and max value
    p_dist = dist + xIndex;
    p_ind = ind + xIndex;
    max_dist = p_dist[0];
    p_ind[0] = 0;

    // Part 1 : sort kth firt elementZ
    for (l = 1; l < k; l++) {
      curr_row = l * width;
      curr_dist = p_dist[curr_row];
      if (curr_dist < max_dist) {
        i = l - 1;
        for (int a = 0; a < l - 1; a++) {
          if (p_dist[a * width] > curr_dist) {
            i = a;
            break;
          }
        }
        for (j = l; j > i; j--) {
          p_dist[j * width] = p_dist[(j - 1) * width];
          p_ind[j * width] = p_ind[(j - 1) * width];
        }
        p_dist[i * width] = curr_dist;
        p_ind[i * width] = l;
      } else {
        p_ind[l * width] = l;
      }
      max_dist = p_dist[curr_row];
    }

    // Part 2 : insert element in the k-th first lines
    max_row = (k - 1) * width;
    for (l = k; l < height; l++) {
      curr_dist = p_dist[l * width];
      if (curr_dist < max_dist) {
        i = k - 1;
        for (int a = 0; a < k - 1; a++) {
          if (p_dist[a * width] > curr_dist) {
            i = a;
            break;
          }
        }
        for (j = k - 1; j > i; j--) {
          p_dist[j * width] = p_dist[(j - 1) * width];
          p_ind[j * width] = p_ind[(j - 1) * width];
        }
        p_dist[i * width] = curr_dist;
        p_ind[i * width] = l;
        max_dist = p_dist[max_row];
      }
    }
  }
}

/**
  * Computes the square root of the first line (width-th first element)
  * of the distance matrix.
  *
  * @param dist    distance matrix
  * @param width   width of the distance matrix
  * @param k       number of neighbors to consider
  */
__global__ void cuParallelSqrt(float *dist, int width, int k) {
  unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
  if (xIndex < width && yIndex < k)
    dist[yIndex * width + xIndex] = sqrt(dist[yIndex * width + xIndex]);
}

//-----------------------------------------------------------------------------------------------//
//                                   K-th NEAREST NEIGHBORS //
//-----------------------------------------------------------------------------------------------//
/**
  * K nearest neighbor algorithm
  * - Initialize CUDA
  * - Allocate device memory
  * - Copy point sets (reference and query points) from host to device memory
  * - Compute the distances + indexes to the k nearest neighbors for each query
 * point
  * - Copy distances from device to host memory
  *
  * @param ref_host      reference points ; pointer to linear matrix
  * @param ref_width     number of reference points ; width of the matrix
  * @param query_host    query points ; pointer to linear matrix
  * @param query_width   number of query points ; width of the matrix
  * @param height        dimension of points ; height of the matrices
  * @param k             number of neighbor to consider
  * @param dist_host     distances to k nearest neighbors ; pointer to linear
 * matrix
  * @param dist_host     indexes of the k nearest neighbors ; pointer to linear
 * matrix
  *
  */
void knn_parallel(float *ref_host, int ref_width, float *query_host,
              int query_width, int height, int k, float *dist_host,
              int *ind_host) {

  unsigned int size_of_float = sizeof(float);
  unsigned int size_of_int = sizeof(int);

  // Variables
  float *query_dev;
  float *ref_dev;
  float *dist_dev;
  int *ind_dev;


  // Allocation of global memory for query points and for distances, CUDA_CHECK
  hipMalloc((void **)&query_dev, query_width * height * size_of_float);
  hipMalloc((void **)&dist_dev, query_width * ref_width * size_of_float);

  // Allocation of global memory for indexes CUDA_CHECK
  hipMalloc((void **)&ind_dev, query_width * k * size_of_int);

  // Allocation of global memory CUDA_CHECK
  hipMalloc((void **)&ref_dev, ref_width * height * size_of_float);

  hipMemcpy(ref_dev, ref_host, ref_width * height * size_of_float,
             hipMemcpyHostToDevice);

  // Copy of part of query actually being treated
  hipMemcpy(query_dev, query_host, query_width * height * size_of_float,
             hipMemcpyHostToDevice);

  // Grids ans threads
  dim3 g_16x16((query_width + 15) / 16, (ref_width + 15) / 16, 1);
  dim3 t_16x16(16, 16, 1);
  //
  dim3 g_256x1((query_width + 255) / 256, 1, 1);
  dim3 t_256x1(256, 1, 1);

  dim3 g_k_16x16((query_width + 15) / 16, (k + 15) / 16, 1);
  dim3 t_k_16x16(16, 16, 1);

  // Kernel 1: Compute all the distances
  cuComputeDistanceGlobal<<<g_16x16, t_16x16>>>(ref_dev, ref_width, query_dev,
                                                query_width, height, dist_dev);

#ifdef DEBUG
  cudaMemcpy(dist_host, dist_dev, query_width * ref_width * size_of_float,
             cudaMemcpyDeviceToHost);

  for (int i = 0; i < query_width * ref_width; i++)
    printf("k1 dist: %d %f\n", i, dist_host[i]);
#endif

  // Kernel 2: Sort each column
  cuInsertionSort<<<g_256x1, t_256x1>>>(dist_dev, ind_dev, query_width,
                                        ref_width, k);

#ifdef DEBUG
  cudaMemcpy(dist_host, dist_dev, query_width * ref_width * size_of_float,
             cudaMemcpyDeviceToHost);

  for (int i = 0; i < query_width * ref_width; i++)
    printf("k2 dist: %d %f\n", i, dist_host[i]);

  cudaMemcpy(ind_host, ind_dev, query_width * k * size_of_int,
             cudaMemcpyDeviceToHost);
  for (int i = 0; i < query_width * k; i++)
    printf("k2 index: %d %d\n", i, ind_host[i]);
#endif

  // Kernel 3: Compute square root of k first elements
  cuParallelSqrt<<<g_k_16x16, t_k_16x16>>>(dist_dev, query_width, k);
  hipDeviceSynchronize();
  // Memory copy of output from device to host
  hipMemcpy(dist_host, dist_dev, query_width * k * size_of_float,
             hipMemcpyDeviceToHost);

  hipMemcpy(ind_host, ind_dev, query_width * k * size_of_int,
             hipMemcpyDeviceToHost);

  // Free memory
  hipFree(ref_dev);
  hipFree(ind_dev);
  hipFree(query_dev);
  hipFree(dist_dev);
}

float compute_distance(const float *ref, int ref_nb, const float *query,
                       int query_nb, int dim, int ref_index, int query_index) {
  float sum = 0.f;
  for (int d = 0; d < dim; ++d) {
    const float diff =
        ref[d * ref_nb + ref_index] - query[d * query_nb + query_index];
    sum += diff * diff;
  }
  return sqrtf(sum);
}

void modified_insertion_sort(float *dist, int *index, int length, int k) {

  // Initialise the first index
  index[0] = 0;

  // Go through all points
  for (int i = 1; i < length; ++i) {

    // Store current distance and associated index
    float curr_dist = dist[i];
    int curr_index = i;

    // Skip the current value if its index is >= k and if it's higher the k-th
    // slready sorted mallest value
    if (i >= k && curr_dist >= dist[k - 1]) {
      continue;
    }

    // Shift values (and indexes) higher that the current distance to the right
    int j = std::min(i, k - 1);
    while (j > 0 && dist[j - 1] > curr_dist) {
      dist[j] = dist[j - 1];
      index[j] = index[j - 1];
      --j;
    }

    // Write the current distance and index at their position
    dist[j] = curr_dist;
    index[j] = curr_index;
  }
}

bool knn_c(const float *ref, int ref_nb, const float *query, int query_nb,
           int dim, int k, float *knn_dist, int *knn_index) {
  // Allocate local array to store all the distances / indexes for a given query
  // point
  float *dist = (float *)malloc(ref_nb * sizeof(float));
  int *index = (int *)malloc(ref_nb * sizeof(int));

  // Allocation checks
  if (!dist || !index) {
    printf("Memory allocation error\n");
    free(dist);
    free(index);
    return false;
  }

  // Process one query point at the time
  for (int i = 0; i < query_nb; ++i) {

    // Compute all distances / indexes
    for (int j = 0; j < ref_nb; ++j) {
      dist[j] = compute_distance(ref, ref_nb, query, query_nb, dim, j, i);
      index[j] = j;
    }

    // Sort distances / indexes
    modified_insertion_sort(dist, index, ref_nb, k);

    // Copy k smallest distances and their associated index
    for (int j = 0; j < k; ++j) {
      knn_dist[j * query_nb + i] = dist[j];
      knn_index[j * query_nb + i] = index[j];
    }
  }

  // Memory clean-up
  free(dist);
  free(index);
  return true;
}

/**
  * Example of use of kNN search CUDA.
  */
int main(void) {
  // Variables and parameters
  float *ref;          // Pointer to reference point array
  float *query;        // Pointer to query point array
  float *dist;         // Pointer to distance array
  int *ind;            // Pointer to index array
  int ref_nb = 4096;   // Reference point number, max=65535
  int query_nb = 4096; // Query point number,     max=65535
  int dim = 68;        // Dimension of points
  int k = 20;          // Nearest neighbors to consider
  int iterations = 100;
  int c_iterations = 1;
  int i;
  const float precision = 0.001f; // distance error max
  int nb_correct_precisions = 0;
  int nb_correct_indexes = 0;
  // Memory allocation
  ref = (float *)malloc(ref_nb * dim * sizeof(float));
  query = (float *)malloc(query_nb * dim * sizeof(float));
  dist = (float *)malloc(query_nb * k * sizeof(float));
  ind = (int *)malloc(query_nb * k * sizeof(float));

  // Init
  srand(2);
  for (i = 0; i < ref_nb * dim; i++)
    ref[i] = (float)rand() / (float)RAND_MAX;
  for (i = 0; i < query_nb * dim; i++)
    query[i] = (float)rand() / (float)RAND_MAX;


  // Display informations
  printf("Number of reference points      : %6d\n", ref_nb);
  printf("Number of query points          : %6d\n", query_nb);
  printf("Dimension of points             : %4d\n", dim);
  printf("Number of neighbors to consider : %4d\n", k);
  printf("Processing kNN search           :\n");

  float *knn_dist = (float *)malloc(query_nb * k * sizeof(float));
  int *knn_index = (int *)malloc(query_nb * k * sizeof(int));
  printf("Ground truth computation in progress...\n\n");
  if (!knn_c(ref, ref_nb, query, query_nb, dim, k, knn_dist, knn_index)) {
    free(ref);
    free(query);
    free(knn_dist);
    free(knn_index);
    return EXIT_FAILURE;
  }

  struct timeval tic;
  struct timeval toc;
  float elapsed_time;

  printf("On CPU: \n");
  gettimeofday(&tic, NULL);
  for (i = 0; i < c_iterations; i++) {
    knn_c(ref, ref_nb, query, query_nb, dim, k, dist, ind);
  }
  gettimeofday(&toc, NULL);
  elapsed_time = toc.tv_sec - tic.tv_sec;
  elapsed_time += (toc.tv_usec - tic.tv_usec) / 1000000.;
  printf(" done in %f s for %d iterations (%f s by iteration)\n", elapsed_time,
         c_iterations, elapsed_time / (c_iterations));

  printf("on GPU: \n");
  gettimeofday(&tic, NULL);
  for (i = 0; i < iterations; i++) {
    knn_parallel(ref, ref_nb, query, query_nb, dim, k, dist, ind);
  }
  gettimeofday(&toc, NULL);
  elapsed_time = toc.tv_sec - tic.tv_sec;
  elapsed_time += (toc.tv_usec - tic.tv_usec) / 1000000.;
  printf(" done in %f s for %d iterations (%f s by iteration)\n", elapsed_time,
         iterations, elapsed_time / (iterations));

  for (int i = 0; i < query_nb * k; ++i) {
    if (fabs(dist[i] - knn_dist[i]) <= precision) {
      nb_correct_precisions++;
    }
    if (ind[i] == knn_index[i]) {
      nb_correct_indexes++;
    } else {
      printf("Mismatch @%d: %d %d\n", i, ind[i], knn_index[i]);
    }
  }

  float precision_accuracy = nb_correct_precisions / ((float)query_nb * k);
  float index_accuracy = nb_correct_indexes / ((float)query_nb * k);
  printf("Precision accuracy %f\nIndex accuracy %f\n", precision_accuracy, index_accuracy);

  free(ind);
  free(dist);
  free(query);
  free(ref);
}
